#include "hip/hip_runtime.h"
//TODO: add license terms
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <algorithm>

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))
#define MAX_PARTITIONS 64
#define WARP_SIZE 64

#define GCN_MFMA_INSTR1 __builtin_amdgcn_mfma_f32_16x16x4f32
#define GCN_MFMA_INSTR __builtin_amdgcn_mfma_f32_4x4x4f16

using floatx4 = __attribute__((__vector_size__(4 * sizeof(float)))) float;
using float16x4 = __attribute__((__vector_size__(4 * sizeof(_Float16)))) _Float16;
typedef float16x4 _Half4;
typedef struct _Half8 { _Half4 xy[2]; } _Half8;
////// Non temporal load stores ///////

#if 1

template <typename T>
__device__ __forceinline__ T load(T* addr) {
  return addr[0];
}

template <typename T>
__device__ __forceinline__ void store(T value, T* addr) {
  addr[0] = value;
}

#else

template <typename T>
__device__ __forceinline__ T load(const T* addr) {
  return __builtin_nontemporal_load(addr);
}

template <>
__device__ __forceinline__
float2 load (const float2* addr) {
  auto addr_alias { reinterpret_cast<const uint64_t *>(addr) };
  auto result = __builtin_nontemporal_load(addr_alias);
  auto ret = reinterpret_cast<float2 *>(&result);
  return ret[0];
}

template <>
__device__ __forceinline__
float4 load (const float4* addr) {
  auto addr_alias { reinterpret_cast<const uint64_t *>(addr) };
  auto result1 = __builtin_nontemporal_load(addr_alias);
  auto result2 = __builtin_nontemporal_load(addr_alias + 1);
  float4 ret{};
  auto ret_alias = reinterpret_cast<float2 *>(&result1);
  ret.x = ret_alias->x;
  ret.y = ret_alias->y;
  ret_alias = reinterpret_cast<float2 *>(&result2);
  ret.z = ret_alias->x;
  ret.w = ret_alias->y;
  return ret;
}

template <>
__device__ __forceinline__
__half load (const __half* addr) {
  auto addr_alias { reinterpret_cast<const uint16_t *>(addr) };
  auto result = __builtin_nontemporal_load(addr_alias);
  auto ret = reinterpret_cast<__half *>(&result);
  return ret[0];
}

template <>
__device__ __forceinline__
__half2 load (const __half2* addr) {
  auto addr_alias { reinterpret_cast<const uint32_t *>(addr) };
  auto result = __builtin_nontemporal_load(addr_alias);
  auto ret = reinterpret_cast<__half2 *>(&result);
  return ret[0];
}

template <>
__device__ __forceinline__
vllm::Half4_ load (const vllm::Half4_* addr) {
  auto addr_alias { reinterpret_cast<const uint64_t *>(addr) };
  auto result = __builtin_nontemporal_load(addr_alias);
  auto ret = reinterpret_cast<vllm::Half4_ *>(&result);
  return ret[0];
}

template <>
__device__ __forceinline__
vllm::Half8_ load (const vllm::Half8_* addr) {
  auto addr_alias { reinterpret_cast<const uint64_t *>(addr) };
  auto result1 = __builtin_nontemporal_load(addr_alias);
  auto result2 = __builtin_nontemporal_load(addr_alias + 1);
  vllm::Half8_ ret {};
  auto ret_alias = reinterpret_cast<vllm::Half4_ *>(&result1);
  ret.x = ret_alias->x;
  ret.y = ret_alias->y;
  ret_alias = reinterpret_cast<vllm::Half4_ *>(&result2);
  ret.z = ret_alias->x;
  ret.w = ret_alias->y;
  return ret;
}

//// Not using nontemporal stores for now
template <typename T>
__device__ __forceinline__ void store(T value, T* addr) {
  return __builtin_nontemporal_store(value, addr);
}

#endif

///////////////////////////////////////

//grid (num_seqs, num_partitions,num_heads/gqa_ratio)
//block (partition size)
template <typename scalar_t, int BLOCK_SIZE, int HEAD_SIZE, int NUM_THREADS, int GQA_RATIO>
__global__ __launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_QKV_kernel(
  const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
  const scalar_t* __restrict__ k_cache,   // [num_blocks, num_kv_heads, head_size/x, block_size, x]
  const scalar_t* __restrict__ v_cache,   // [num_blocks, num_kv_heads, head_size, block_size]
  const int num_kv_heads,
  const float scale,
  const int* __restrict__ block_tables,   // [num_seqs, max_num_blocks_per_seq]
  const int* __restrict__ context_lens,   // [num_seqs]
  const int max_num_blocks_per_seq,
  const float* __restrict__ alibi_slopes, // [num_heads]
  const int q_stride,
  const int kv_block_stride,
  const int kv_head_stride,
  float* __restrict__ exp_sums,           // [num_seqs, num_heads, max_num_partitions]
  float* __restrict__ max_logits,         // [num_seqs, num_heads, max_num_partitions]
  scalar_t* __restrict__ out,             // [num_seqs, num_heads, max_num_partitions, head_size]
  scalar_t* __restrict__ final_out,             // [num_seqs, num_heads, head_size]
#if 0
  scalar_t* __restrict__ qk_out,             // [num_heads, num_seqs, max_ctx_blocks,block_size]
#endif
  int max_ctx_blocks
  ) {
      constexpr int NWARPS = NUM_THREADS/WARP_SIZE;
      const int warpid = threadIdx.x / WARP_SIZE;
      const int laneid = threadIdx.x % WARP_SIZE;
      const int lane4id = laneid%4;

      const int seq_idx = blockIdx.x;
      const int partition_idx = blockIdx.y;
      const int partition_size = blockDim.x;
      const int max_num_partitions = gridDim.y;

      const int context_len = context_lens[seq_idx];
      const int partition_start_token_idx = partition_idx * partition_size;
      //exit if partition is out of context for seq
      if (partition_start_token_idx >= context_len) {
          return;
      }
      constexpr int QHLOOP = DIVIDE_ROUND_UP(GQA_RATIO,4); // each 4 lanes fetch 4 different qheads, total qheads =8, so qhloop is 2
      constexpr int GQA_RATIO4 = 4*QHLOOP;
      __shared__ float shared_qk_max[NWARPS][GQA_RATIO4+1];
      __shared__ float shared_exp_sum[NWARPS][GQA_RATIO4+1];
      _Half8 Qlocal[QHLOOP];
      constexpr int x = 16 / sizeof(scalar_t);
      constexpr int HELOOP = HEAD_SIZE/x;
      _Half8 Klocal[HELOOP];
      constexpr int VHLOOP = HEAD_SIZE/WARP_SIZE; //v head_size dimension is distributed across lanes
      constexpr int VTLOOP = 8; //16 separate 4xtokens across warp -> 16/2 8xtokens
      _Half8 Vlocal[VHLOOP][VTLOOP];
      floatx4 dout[QHLOOP];
      float qk_max[QHLOOP];
      #pragma unroll
      for (int h=0; h<QHLOOP; h++) {
        dout[h] = {0};
        qk_max[h] = -FLT_MAX;
      }

      const int wg_start_head_idx = blockIdx.z * GQA_RATIO;
      const int wg_start_kv_head_idx = blockIdx.z;

      const int warp_start_token_idx = partition_start_token_idx + warpid*WARP_SIZE;

      if (warp_start_token_idx  >= context_len) { //warp out of context
        #pragma unroll
        for(int h=0;h<GQA_RATIO4;h++) {
            shared_qk_max[warpid][h] = -FLT_MAX;
            shared_exp_sum[warpid][h] = 0.0f;
        }
      }
      else {//warp within context

      const int num_context_blocks = DIVIDE_ROUND_UP(context_len, BLOCK_SIZE);
      const int last_ctx_block = num_context_blocks - 1;

      const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;

      const int local_token_idx = threadIdx.x;
      const int global_token_idx = partition_start_token_idx + local_token_idx;

      const int block_idx = (global_token_idx < context_len) ? global_token_idx / BLOCK_SIZE : last_ctx_block;

      const int physical_block_number = block_table[block_idx];

      //constexpr int HELOOP = HEAD_SIZE/8;
      //each 4 lanes fetch 8 helems, so warp fetches 8*16 = 128 helems


      const scalar_t* q_ptr = q + seq_idx*q_stride + wg_start_head_idx*HEAD_SIZE;
      const _Half8*  q_ptrh8 = reinterpret_cast<const _Half8 *>(q_ptr);
      const int qhead_elemh8 = laneid/4;
      #pragma unroll
      for (int h=0; h<QHLOOP-1; h++) {
          const int qhead_idx = h*4 + lane4id;
          Qlocal[h] = q_ptrh8[qhead_idx*HEAD_SIZE/8 + qhead_elemh8];
      }
      const int final_qhead_idx = 4*(QHLOOP-1) + lane4id;
      if (final_qhead_idx < GQA_RATIO) {
          Qlocal[QHLOOP-1] = q_ptrh8[final_qhead_idx*HEAD_SIZE/8 + qhead_elemh8];
      } else {
          Qlocal[QHLOOP-1].xy[0] = {0};
          Qlocal[QHLOOP-1].xy[1] = {0};
      }

      //const int kv_head_idx = 0;
      const scalar_t* k_ptr = k_cache + physical_block_number * kv_block_stride
                                        + wg_start_kv_head_idx * kv_head_stride;
      const _Half8* k_ptrh8 = reinterpret_cast<const _Half8 *>(k_ptr);

      const int physical_block_offset = local_token_idx%BLOCK_SIZE; //since x=half8, physical_block_offset is already cast as _H8


      #pragma unroll
      for (int d=0;d<HELOOP;d++) {
       Klocal[d] = k_ptrh8[d*BLOCK_SIZE + physical_block_offset];
      }

      float alibi_slope[QHLOOP];
      if (alibi_slopes != nullptr) {
        #pragma unroll
        for (int h=0; h<QHLOOP; h++) {
            const int qhead_idx = h*4 + lane4id;
            alibi_slope[h] = (qhead_idx < GQA_RATIO) ? alibi_slopes[wg_start_head_idx + qhead_idx] : 0.f;
        }
      }

#if 1

      const scalar_t* v_ptr = v_cache + wg_start_kv_head_idx * kv_head_stride;
      const _Half8* v_ptrh8 = reinterpret_cast<const _Half8*>(v_ptr);
      const int warp_start_block_idx = warp_start_token_idx/BLOCK_SIZE;
      //iterate over each v block
      #pragma unroll
      for (int b=0;b<8*VTLOOP/BLOCK_SIZE;b++) {
	    const int vblock_idx = warp_start_block_idx + b;
        const int vblock_idx_ctx = (vblock_idx <= last_ctx_block) ? vblock_idx : last_ctx_block;
        const int vphysical_block_number = block_table[vblock_idx_ctx];
      	const _Half8* v_ptrh8b = v_ptrh8 + (vphysical_block_number * kv_block_stride)/8;
        //iterate over each head elem (within head_size)
        #pragma unroll
        for (int h=0;h<VHLOOP;h++) {
            const int head_size_elem = h*WARP_SIZE + laneid;
            const _Half8* v_ptrh8be = v_ptrh8b + head_size_elem*BLOCK_SIZE/8;
            //iterate over all velems within block
            #pragma unroll
            for (int d=0;d<BLOCK_SIZE/8;d++) {
                Vlocal[h][b*BLOCK_SIZE/8+d] = v_ptrh8be[d];
            }
        }
      }
#endif

      //dout[0] = {0};
      //dout[1] = {0};

        #pragma unroll
        for (int h=0;h<QHLOOP;h++) {
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[0].xy[0], dout[h], 4, 0, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[0].xy[1], dout[h], 4, 0, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[1].xy[0], dout[h], 4, 1, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[1].xy[1], dout[h], 4, 1, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[2].xy[0], dout[h], 4, 2, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[2].xy[1], dout[h], 4, 2, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[3].xy[0], dout[h], 4, 3, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[3].xy[1], dout[h], 4, 3, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[4].xy[0], dout[h], 4, 4, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[4].xy[1], dout[h], 4, 4, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[5].xy[0], dout[h], 4, 5, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[5].xy[1], dout[h], 4, 5, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[6].xy[0], dout[h], 4, 6, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[6].xy[1], dout[h], 4, 6, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[7].xy[0], dout[h], 4, 7, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[7].xy[1], dout[h], 4, 7, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[8].xy[0], dout[h], 4, 8, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[8].xy[1], dout[h], 4, 8, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[9].xy[0], dout[h], 4, 9, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[9].xy[1], dout[h], 4, 9, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[10].xy[0], dout[h], 4, 10, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[10].xy[1], dout[h], 4, 10, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[11].xy[0], dout[h], 4, 11, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[11].xy[1], dout[h], 4, 11, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[12].xy[0], dout[h], 4, 12, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[12].xy[1], dout[h], 4, 12, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[13].xy[0], dout[h], 4, 13, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[13].xy[1], dout[h], 4, 13, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[14].xy[0], dout[h], 4, 14, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[14].xy[1], dout[h], 4, 14, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[0], Klocal[15].xy[0], dout[h], 4, 15, 0);
          dout[h] = GCN_MFMA_INSTR(Qlocal[h].xy[1], Klocal[15].xy[1], dout[h], 4, 15, 0);
          dout[h]*=scale;
        }
        //transpose dout so that 4 token ids are in each lane, and 4 heads are across 4 lanes
        #pragma unroll
        for (int h=0;h<QHLOOP;h++) {
            floatx4 tmp={0};
            #pragma unroll
            for(int i=0; i<4; i++) {
                const float B = (lane4id==i)? 1.0f : 0.0f;
                //const float A = (global_token_idx < context_len) ? dout[h][i] : 0.0f;
                tmp = __builtin_amdgcn_mfma_f32_4x4x1f32(dout[h][i], B, tmp, 0, 0, 0);
                //tmp = __builtin_amdgcn_mfma_f32_4x4x1f32(A, B, tmp, 0, 0, 0);
            }
            dout[h] = tmp;
        }

        const int lane4_token_idx = 4*(global_token_idx>>2);
        const int alibi_offset = lane4_token_idx - context_len + 1;
        if (alibi_slopes != nullptr) {
          #pragma unroll
          for (int h=0;h<QHLOOP;h++) {
              #pragma unroll
              for(int i=0; i<4; i++) {
                  dout[h][i] += alibi_slope[h] * (alibi_offset + i);
              }
          }
        }

        #pragma unroll
        for (int h=0;h<QHLOOP;h++) {
            qk_max[h] = -FLT_MAX;
            #pragma unroll
            for(int i=0; i<4; i++) {
                qk_max[h] = (lane4_token_idx+i < context_len) ? fmaxf(qk_max[h], dout[h][i]) : qk_max[h];
            }
            #pragma unroll
            for(int mask=WARP_SIZE/2; mask>=4; mask/=2) {
                qk_max[h] = fmaxf(qk_max[h], __shfl_xor(qk_max[h],mask));
            }
        }

        float exp_sum[QHLOOP];
        #pragma unroll
        for (int h=0;h<QHLOOP;h++) {
            exp_sum[h] = 0.0f;
            #pragma unroll
            for(int i=0; i<4; i++) {
                dout[h][i] = (lane4_token_idx+i < context_len) ? __expf(dout[h][i] - qk_max[h]) : 0.0f;
                exp_sum[h] += dout[h][i];
            }
            #pragma unroll
            for(int mask=WARP_SIZE/2; mask>=4; mask/=2) {
                exp_sum[h] += __shfl_xor(exp_sum[h],mask);
            }
        }


        #pragma unroll
        for (int h=0;h<QHLOOP;h++) {
            const int head_idx = 4*h+lane4id;
            shared_qk_max[warpid][head_idx] = qk_max[h];
            shared_exp_sum[warpid][head_idx] = exp_sum[h];
        }
    }//warp within context

        __syncthreads();

    //float global_qk_max[QHLOOP];
    //float global_inv_sum_scale[QHLOOP];
    //float global_exp_scale[QHLOOP];
    const int num_heads = gridDim.z*GQA_RATIO;
    float* max_logits_ptr = max_logits + seq_idx * num_heads * max_num_partitions
                                       + partition_idx;
    float* exp_sums_ptr = exp_sums + seq_idx * num_heads * max_num_partitions
                                   + partition_idx;
    #pragma unroll
    for (int h=0;h<QHLOOP;h++) {
        float global_qk_max = -FLT_MAX;
        float warp_qk_max[NWARPS];
        const int head_idx = 4*h+lane4id;
        #pragma unroll
        for (int w=0; w<NWARPS; w++) {
            warp_qk_max[w] = shared_qk_max[w][head_idx];
            global_qk_max = fmaxf(global_qk_max,warp_qk_max[w]);
        }
        //global_exp_scale[h] = __expf(qk_max[h] - global_qk_max);
        float global_exp_sum = 0.0f;
        #pragma unroll
        for (int w=0; w<NWARPS; w++) {
            global_exp_sum += shared_exp_sum[w][head_idx] * __expf(warp_qk_max[w] - global_qk_max);
        }
        if (head_idx < GQA_RATIO) {
          max_logits_ptr[(wg_start_head_idx + head_idx) * max_num_partitions] = global_qk_max;
          exp_sums_ptr[(wg_start_head_idx + head_idx) * max_num_partitions] = global_exp_sum;
        }
        //global_inv_sum[h] = 1.0f/global_exp_sum;
        const float global_inv_sum_scale = __fdividef(1.f, global_exp_sum + 1e-6f) * __expf(qk_max[h] - global_qk_max);
        dout[h] *= global_inv_sum_scale;
    }
    //logits[h] -> every 4 lanes hold 4 heads, each lane holds 4 tokens, there are 4x16 tokens across warp
    float16x4 logits[QHLOOP];
    #pragma unroll
    for (int h=0;h<QHLOOP;h++) {
        #pragma unroll
        for (int i=0;i<4;i++) {
            logits[h][i] = (scalar_t) dout[h][i];
        }
    }

    //float16x4 vout[QHLOOP][VHLOOP];
    __shared__ float16x4 vout_shared[QHLOOP][VHLOOP][WARP_SIZE][NWARPS+1];

    if (warp_start_token_idx  >= context_len) { //warp out of context
    #pragma unroll
    for (int qh=0; qh<QHLOOP; qh++) {
        #pragma unroll
        for (int vh=0; vh<VHLOOP; vh++) {
            vout_shared[qh][vh][laneid][warpid] = {0};
        }
    }
    }
    else{//warp in context
    //iterate across heads
    #pragma unroll
    for (int qh=0; qh<QHLOOP; qh++) {
        //iterate over each v head elem (within head_size)
        #pragma unroll
        for (int vh=0; vh<VHLOOP; vh++) {
            floatx4 acc = {0};
            //iterate over tokens
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][0].xy[0], acc, 4, 0, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][0].xy[1], acc, 4, 1, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][1].xy[0], acc, 4, 2, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][1].xy[1], acc, 4, 3, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][2].xy[0], acc, 4, 4, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][2].xy[1], acc, 4, 5, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][3].xy[0], acc, 4, 6, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][3].xy[1], acc, 4, 7, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][4].xy[0], acc, 4, 8, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][4].xy[1], acc, 4, 9, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][5].xy[0], acc, 4, 10, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][5].xy[1], acc, 4, 11, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][6].xy[0], acc, 4, 12, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][6].xy[1], acc, 4, 13, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][7].xy[0], acc, 4, 14, 0);
            acc = GCN_MFMA_INSTR(logits[qh], Vlocal[vh][7].xy[1], acc, 4, 15, 0);
            float16x4 tmp;
            #pragma unroll
            for(int i=0; i<4; i++) {
                tmp[i] = (scalar_t) acc[i];
            }
            vout_shared[qh][vh][laneid][warpid] = tmp;
        }
    }
    }//warp in context

    __syncthreads();

    if (warpid==0) {
        float16x4 vout[QHLOOP][VHLOOP];
        //iterate across heads
        //scalar_t* out_ptr = out + seq_idx*num_heads*HEAD_SIZE;
        scalar_t* out_ptr;
        int out_num_partitions;
        if (context_len > partition_size) {
            out_num_partitions = max_num_partitions;
            out_ptr = out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE
                            + partition_idx * HEAD_SIZE;
        } else {
            out_num_partitions = 1;
            out_ptr = final_out + seq_idx * num_heads * HEAD_SIZE;
        }
        #pragma unroll
        for (int qh=0; qh<QHLOOP; qh++) {
            //iterate over each v head elem (within head_size)
            #pragma unroll
            for (int vh=0; vh<VHLOOP; vh++) {
                vout[qh][vh] = {0};
                #pragma unroll
                for (int w=0; w<NWARPS; w++) {
                    vout[qh][vh] += vout_shared[qh][vh][laneid][w];
                }
                const int head_size_elem = vh*WARP_SIZE + laneid;
                #pragma unroll
                for (int i=0; i<4; i++) {
                    const int head_idx = 4*qh + i;
                    //out_ptr[head_idx*HEAD_SIZE + head_size_elem] = vout[qh][vh][i];
                    if (head_idx < GQA_RATIO) {
                      //out_ptr[(wg_start_head_idx + head_idx) * max_num_partitions * HEAD_SIZE + head_size_elem] = vout[qh][vh][i];
                      out_ptr[(wg_start_head_idx + head_idx) * out_num_partitions * HEAD_SIZE + head_size_elem] = vout[qh][vh][i];
                    }
                }
            }
        }
    }

#if 0
    const int num_seqs = gridDim.x;
    const int global_token4id = global_token_idx/4;
    #pragma unroll
    for (int t=0;t<4;t++) {
        #pragma unroll
        for (int h=0;h<QHLOOP;h++) {
          //const int head_idx = h*4 + t;
          const int head_idx = h*4 + lane4id;
	      //qk_out[head_idx*num_seqs*max_ctx_blocks*BLOCK_SIZE + seq_idx*max_ctx_blocks*BLOCK_SIZE + global_token_idx] = (scalar_t)dout[h][t];
	       qk_out[head_idx*num_seqs*max_ctx_blocks*BLOCK_SIZE + seq_idx*max_ctx_blocks*BLOCK_SIZE + 4*global_token4id + t] = logits[h][t];
	      //qk_out[head_idx*num_seqs*max_ctx_blocks*BLOCK_SIZE + seq_idx*max_ctx_blocks*BLOCK_SIZE + 4*global_token4id + t] = vout[h][t%2][t];
        }
    }
#endif

  }

// Grid: (num_heads, num_seqs).
template<
  typename scalar_t,
  int HEAD_SIZE,
  int NUM_THREADS,
  int PARTITION_SIZE>
__global__ __launch_bounds__(NUM_THREADS) void paged_attention_ll4mi_reduce_kernel(
  scalar_t* __restrict__ out,             // [num_seqs, num_heads, head_size]
  const float* __restrict__ exp_sums,     // [num_seqs, num_heads, max_num_partitions]
  const float* __restrict__ max_logits,   // [num_seqs, num_heads, max_num_partitions]
  const scalar_t* __restrict__ tmp_out,   // [num_seqs, num_heads, max_num_partitions, head_size]
  const int* __restrict__ context_lens,   // [num_seqs]
  const int max_num_partitions) {
  const int num_heads = gridDim.x;
  const int head_idx = blockIdx.x;
  const int seq_idx = blockIdx.y;
  const int context_len = context_lens[seq_idx];
  const int num_partitions = DIVIDE_ROUND_UP(context_len, PARTITION_SIZE);
  if (num_partitions == 1) {
    // No need to reduce. Only copy tmp_out to out.
    //scalar_t* out_ptr = out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
    //const scalar_t* tmp_out_ptr = tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE
    //                                      + head_idx * max_num_partitions * HEAD_SIZE;
    //for (int i = threadIdx.x; i < HEAD_SIZE; i += blockDim.x) {
    //  out_ptr[i] = tmp_out_ptr[i];
    //}
    // Terminate the thread block.
    //if num_partitions==1, main kernel will write to out directly, no work in reduction kernel
    return;
  }

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int warp_idx = threadIdx.x / WARP_SIZE;
  const int lane = threadIdx.x % WARP_SIZE;

  // Size: 2 * num_partitions.
  extern __shared__ char shared_mem[];
  // Workspace for reduction.
  __shared__ float red_smem[2 * NUM_WARPS];
  __shared__ float shared_global_exp_sum;
  //float reg_max_logits[MAX_PARTITIONS]; //dependent on max_num_partitions: assume 32K max context div 1K Partition size -> TODO: make this proper template param
  //Assume code below is optimized for MAX_PARTITIONS<=64 TODO: handle larger than warp size cases later
  float* shared_max_logits = reinterpret_cast<float*>(shared_mem);
  float* shared_exp_sums = reinterpret_cast<float*>(shared_mem + sizeof(float) * num_partitions);
  //scalar_t tmp_outs[MAX_PARTITIONS];

  // Load max logits to shared memory.
  const float* max_logits_ptr = max_logits + seq_idx * num_heads * max_num_partitions
                                           + head_idx * max_num_partitions;
  ////float max_logit = -FLT_MAX;
  //for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
  ////for (int i = threadIdx.x; i < MAX_PARTITIONS; i += blockDim.x) {
    //const float l = max_logits_ptr[i];
    //shared_max_logits[i] = l;
    ////reg_max_logits[i] =  max_logits_ptr[i];  //TODO: review this -> right now num_partitions is very small <=32
    //max_logit = fmaxf(max_logit, l);
    ////max_logit = fmaxf(max_logit, reg_max_logits[i]);
  ////}
  //__syncthreads();
  float max_logit = (threadIdx.x < num_partitions) ? max_logits_ptr[threadIdx.x]:-FLT_MAX;
  float reg_max_logit =  max_logit;

  // Get the global max logit.
  // Reduce within the warp.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    max_logit = fmaxf(max_logit, __shfl_xor(max_logit, mask));
  }
//  if (lane == 0) {
//    red_smem[warp_idx] = max_logit;
//  }

//  if (num_partitions >= WARP_SIZE) {
//  __syncthreads();
//  // Reduce across warps.
//  max_logit = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
//#pragma unroll
//  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
//    max_logit = fmaxf(max_logit, __shfl_xor(max_logit, mask));
//  }
//  // Broadcast the max value to all threads.
//  //max_logit = __shfl(max_logit, 0);
//  }
  // Load rescaled exp sums to shared memory.
  const float* exp_sums_ptr = exp_sums + seq_idx * num_heads * max_num_partitions
                                       + head_idx * max_num_partitions;
  float global_exp_sum = 0.0f;

  //for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
  //  //float l = shared_max_logits[i];
  //  //float l = reg_max_logits[i];
  //  float rescaled_exp_sum = exp_sums_ptr[i] * expf(reg_max_logits[i] - max_logit);
  //  global_exp_sum += rescaled_exp_sum;
  //  shared_exp_sums[i] = rescaled_exp_sum;
  //}
  float rescaled_exp_sum = (threadIdx.x < num_partitions) ? exp_sums_ptr[threadIdx.x] * expf(reg_max_logit - max_logit) : 0.0f;
  global_exp_sum += rescaled_exp_sum;
  //if (threadIdx.x < num_partitions) {
    //shared_exp_sums[threadIdx.x] = (threadIdx.x < num_partitions) ? rescaled_exp_sum : 0.0f;
    shared_exp_sums[threadIdx.x] = rescaled_exp_sum;
  //}

#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    global_exp_sum += __shfl_xor(global_exp_sum, mask);
  }
  if (threadIdx.x==0) {
    shared_global_exp_sum = global_exp_sum;
  }
  __syncthreads();

  //global_exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], global_exp_sum);
  const float inv_global_exp_sum = __fdividef(1.0f, shared_global_exp_sum + 1e-6f);

  // Aggregate tmp_out to out.
  scalar_t* out_ptr = out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
    const scalar_t* tmp_out_ptr = tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE
                                        + head_idx * max_num_partitions * HEAD_SIZE;
//#pragma unroll
  //for (int i = threadIdx.x; i < HEAD_SIZE; i += NUM_THREADS) {
  //if (threadIdx.x < HEAD_SIZE) { //TODO: assume HEAD_SIZE < NUM_THREADS, revisit this assumption later
    constexpr int MAX_NPAR = 64;
    scalar_t tmps[MAX_NPAR];
    int lastj=0;
    #pragma unroll
    for (int j = 0; j < MAX_NPAR; j++) {
        lastj = (j<num_partitions) ? j: lastj;
        tmps[j] = tmp_out_ptr[lastj * HEAD_SIZE + threadIdx.x];
    }

    //float tmpf[64];
    //#pragma unroll
    //for (int j = 0; j < 64; j++) {
    //    const float mult = (j<num_partitions) ? 1.0f: 0.0f;
    //    tmpf[j] = (float)tmps[j] * mult;
    //}

    float acc = 0.0f;
    //for (int j = 0; j < num_partitions; ++j) {
    #pragma unroll
    for (int j = 0; j < MAX_NPAR; j++) {
      //acc += to_float(tmp_out_ptr[j * HEAD_SIZE + i]) * shared_exp_sums[j] * inv_global_exp_sum;
      const float expsum = (j<num_partitions) ? shared_exp_sums[j] : 0.0f;
      //acc += (float)tmp_out_ptr[j * HEAD_SIZE + threadIdx.x] * shared_exp_sums[j] * inv_global_exp_sum;
      acc += (float)tmps[j] * expsum;
    }
    acc *= inv_global_exp_sum;
    //from_float(out_ptr[threadIdx.x], acc);
    out_ptr[threadIdx.x] = (scalar_t)acc;
  //}
}

#define CALL_CUSTOM_LAUNCHER(T)                             \
  paged_attention_custom_launcher<T>(                       \
    out,                                                            \
    exp_sums,                                                       \
    max_logits,                                                     \
    tmp_out,                                                        \
    query,                                                          \
    key_cache,                                                      \
    value_cache,                                                    \
    num_kv_heads,                                                   \
    scale,                                                          \
    block_tables,                                                   \
    context_lens,                                                   \
    max_context_len,\
    alibi_slopes);

#define LAUNCH_CUSTOM_ATTENTION(GQA_RATIO)                                                  \
  paged_attention_ll4mi_QKV_kernel<T,BLOCK_SIZE,HEAD_SIZE,NTHR,GQA_RATIO>      \
  <<<grid, block, 0, stream>>>(                                                 \
    query_ptr,                                                                                \
    key_cache_ptr,                                                                            \
    value_cache_ptr,                                                                          \
    num_kv_heads,                                                                         \
    scale,                                                                                    \
    block_tables_ptr,                                                                         \
    context_lens_ptr,                                                                         \
    max_num_blocks_per_seq,                                                                   \
    alibi_slopes_ptr,                                                                         \
    q_stride,                                                                                 \
    kv_block_stride,                                                                          \
    kv_head_stride, exp_sums_ptr, max_logits_ptr, tmp_out_ptr,out_ptr,max_ctx_blocks);

template<typename T, int BLOCK_SIZE=16, int HEAD_SIZE=128>
void paged_attention_custom_launcher(
  torch::Tensor& out,
  torch::Tensor& exp_sums,
  torch::Tensor& max_logits,
  torch::Tensor& tmp_out,
  torch::Tensor& query,
  torch::Tensor& key_cache,
  torch::Tensor& value_cache,
  const int num_kv_heads,
  float scale,
  torch::Tensor& block_tables,
  torch::Tensor& context_lens,
  int max_context_len,
#if 0
  torch::Tensor& qk_out,
  torch::Tensor& softmax_out,
#endif
  const c10::optional<torch::Tensor>& alibi_slopes) {

  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  //int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  //assert(head_size % thread_group_size == 0);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr = alibi_slopes ?
    reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
    : nullptr;

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  float* exp_sums_ptr = reinterpret_cast<float*>(exp_sums.data_ptr());
  float* max_logits_ptr = reinterpret_cast<float*>(max_logits.data_ptr());
  T* tmp_out_ptr = reinterpret_cast<T*>(tmp_out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  T* key_cache_ptr = reinterpret_cast<T*>(key_cache.data_ptr());
  T* value_cache_ptr = reinterpret_cast<T*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* context_lens_ptr = context_lens.data_ptr<int>();
#if 0
  T* qk_out_ptr = reinterpret_cast<T*>(qk_out.data_ptr());
  T* softmax_out_ptr = reinterpret_cast<T*>(softmax_out.data_ptr());
#endif
  //constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  //int logits_size = PARTITION_SIZE * sizeof(float);
  //int outputs_size = (NUM_WARPS / 2) * head_size * sizeof(float);

  // For paged attention v2 kernel.
  //dim3 grid(num_heads, num_seqs, max_num_partitions);
  //int shared_mem_size = std::max(logits_size, outputs_size);
  //// For paged attention v2 reduce kernel.
  //assert(max_num_partitions <= MAX_PARTITIONS);
  //assert(MAX_PARTITIONS<=head_size);
  //dim3 reduce_grid(num_heads, num_seqs);
  //dim3 reduce_block(head_size); //TODO: assumes max_partitions < head_SIZE
  ////dim3 reduce_block(NUM_THREADS);
  //int reduce_shared_mem_size = 2 * max_num_partitions * sizeof(float);

  int max_ctx_blocks = DIVIDE_ROUND_UP(max_context_len, BLOCK_SIZE);

  //dim3 grid(num_seqs,BLOCK_RATIO_PER_WG*max_ctx_blocks);
  //dim3 block(num_heads*HEAD_SIZE*sizeof(T)/sizeof(float4));
  constexpr int NTHR = 256;
  const int max_num_partitions = DIVIDE_ROUND_UP(max_context_len, NTHR);
  //constexpr int NPAR = 2;
  //constexpr int GQA_RATIO = 32;
  const int gqa_ratio = num_heads/num_kv_heads;
  //assert(gqa_ratio>=4);
  //assert(gqa_ratio%4==0);
  assert(num_heads%num_kv_heads==0);
  assert(head_size==HEAD_SIZE);
  dim3 grid(num_seqs,max_num_partitions,num_kv_heads);
  dim3 block(NTHR);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  switch (gqa_ratio) {
      case 1: LAUNCH_CUSTOM_ATTENTION(1); break;
      case 2: LAUNCH_CUSTOM_ATTENTION(2); break;
      case 3: LAUNCH_CUSTOM_ATTENTION(3); break;
      case 4: LAUNCH_CUSTOM_ATTENTION(4); break;
      case 5: LAUNCH_CUSTOM_ATTENTION(5); break;
      case 6: LAUNCH_CUSTOM_ATTENTION(6); break;
      case 7: LAUNCH_CUSTOM_ATTENTION(7); break;
      case 8: LAUNCH_CUSTOM_ATTENTION(8); break;
      case 9: LAUNCH_CUSTOM_ATTENTION(9); break;
      case 10: LAUNCH_CUSTOM_ATTENTION(10); break;
      case 11: LAUNCH_CUSTOM_ATTENTION(11); break;
      case 12: LAUNCH_CUSTOM_ATTENTION(12); break;
      case 13: LAUNCH_CUSTOM_ATTENTION(13); break;
      case 14: LAUNCH_CUSTOM_ATTENTION(14); break;
      case 15: LAUNCH_CUSTOM_ATTENTION(15); break;
      case 16: LAUNCH_CUSTOM_ATTENTION(16); break;
      default:
        TORCH_CHECK(false, "Unsupported gqa ratio: ", gqa_ratio);
        break;
  }
  //dim3 grid2(num_heads,num_seqs,head_size/HEAD_ELEMS_PER_WG);
  //dim3 block2(1024);
  // LAUNCH_CUSTOM_ATTENTION2;
  //constexpr int PARSIZE = 256;
  dim3 reduce_grid(num_heads, num_seqs);
  dim3 reduce_block(head_size); //TODO: assumes max_partitions < head_SIZE
  int reduce_shared_mem_size = 2 * max_num_partitions * sizeof(float);
  paged_attention_ll4mi_reduce_kernel<T, HEAD_SIZE, HEAD_SIZE, NTHR>
  <<<reduce_grid, reduce_block, reduce_shared_mem_size, stream>>>(
    out_ptr,
    exp_sums_ptr,
    max_logits_ptr,
    tmp_out_ptr,
    context_lens_ptr,
    max_num_partitions);
  //switch (head_size) {
  //  // NOTE(woosuk): To reduce the compilation time, we only compile for the
  //  // head sizes that we use in the model. However, we can easily extend this
  //  // to support any head size which is a multiple of 16.
  //  case 64:
  //    LAUNCH_PAGED_ATTENTION_V2(64);
  //    break;
  //  case 80:
  //    LAUNCH_PAGED_ATTENTION_V2(80);
  //    break;
  //  case 96:
  //    LAUNCH_PAGED_ATTENTION_V2(96);
  //    break;
  //  case 112:
  //    LAUNCH_PAGED_ATTENTION_V2(112);
  //    break;
  //  case 128:
  //    LAUNCH_PAGED_ATTENTION_V2(128);
  //    break;
  //  case 256:
  //    LAUNCH_PAGED_ATTENTION_V2(256);
  //    break;
  //  default:
  //    TORCH_CHECK(false, "Unsupported head size: ", head_size);
  //    break;
  //}
}

void paged_attention_custom(
  torch::Tensor& out,             // [num_seqs, num_heads, head_size]
  torch::Tensor& exp_sums,        // [num_seqs, num_heads, max_num_partitions]
  torch::Tensor& max_logits,      // [num_seqs, num_heads, max_num_partitions]
  torch::Tensor& tmp_out,         // [num_seqs, num_heads, max_num_partitions, head_size]
  torch::Tensor& query,           // [num_seqs, num_heads, head_size]
  torch::Tensor& key_cache,       // [num_blocks, num_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,     // [num_blocks, num_heads, head_size, block_size]
  int num_kv_heads,
  float scale,
  torch::Tensor& block_tables,    // [num_seqs, max_num_blocks_per_seq]
  torch::Tensor& context_lens,    // [num_seqs]
  int block_size,
  int max_context_len,
#if 0
  torch::Tensor& qk_out,
  torch::Tensor& softmax_out,
#endif
  const c10::optional<torch::Tensor>& alibi_slopes,
  const std::string& kv_cache_dtype) {
  assert(block_size==16);
  if (query.dtype() == at::ScalarType::Half) {
    //CALL_V2_LAUNCHER_BLOCK_SIZE(__half);
    CALL_CUSTOM_LAUNCHER(_Float16);
  } else {
    TORCH_CHECK(false, "Unsupported data type: ", query.dtype());
  }
}

#undef WARP_SIZE
#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP
