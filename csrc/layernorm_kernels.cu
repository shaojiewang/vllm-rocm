#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "dispatch_utils.h"
#include "reduction_utils.cuh"
#include "attention/dtype_float16.cuh"

namespace vllm {

// TODO(woosuk): Further optimize this kernel.
template<typename scalar_t>
__global__ void rms_norm_kernel(
  scalar_t* __restrict__ out,             // [..., hidden_size]
  const scalar_t* __restrict__ input,     // [..., hidden_size]
  const scalar_t* __restrict__ weight,    // [hidden_size]
  const float epsilon,
  const int num_tokens,
  const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    const float x = (float) input[blockIdx.x * hidden_size + idx];
    variance += x * x;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float) input[blockIdx.x * hidden_size + idx];
    out[blockIdx.x * hidden_size + idx] = ((scalar_t) (x * s_variance)) * weight[idx];
  }
}


/* Helper POD struct to generate vectorized and packed FP16 ops
   for appropriate overloads of fused_add_rms_norm_kernel.
   Only special member functions and functions that are necessary
   in that kernel are implemented.
 */
template<int width>
struct _halfVec {
  /* Not theoretically necessary that width is a power of 2 but should 
     almost always be the case for optimization purposes */ 
  static_assert(width > 0 && (width & (width - 1)) == 0,
                "Width is not a positive power of 2!");
  __half data[width];

  __device__ _halfVec() = default;
  __device__ ~_halfVec() = default;
  __device__ _halfVec(const _halfVec<width>&) = default;
  __device__ _halfVec& operator=(const _halfVec<width>&) = default;
  __device__ _halfVec(_halfVec<width>&&) = default;
  __device__ _halfVec& operator=(_halfVec<width>&&) = default;

  __device__ inline _halfVec& operator+=(const _halfVec<width>& other) {
    if constexpr (width % 2 == 0) {
      for (int i = 0; i < width; i += 2) {
        __half2 z = __half2{data[i], data[i+1]};
        z += __half2{other.data[i], other.data[i+1]};
        data[i] = z.x;
        data[i+1] = z.y;
      }
    } else {
      #pragma unroll
      for (int i = 0; i < width; ++i)
        data[i] += other.data[i];
    }
    return *this;
  }

  __device__ inline _halfVec& operator*=(const _halfVec<width>& other) {
    if constexpr (width % 2 == 0) {
      for (int i = 0; i < width; i += 2) {
        __half2 z = __half2{data[i], data[i+1]};
        z *= __half2{other.data[i], other.data[i+1]};
        data[i] = z.x;
        data[i+1] = z.y;
      }
    } else {
      #pragma unroll
      for (int i = 0; i < width; ++i)
        data[i] *= other.data[i];
    }
    return *this;
  }

  __device__ inline _halfVec& operator*=(const float scale) {
    if constexpr (width % 2 == 0) {
      #pragma unroll
      for (int i = 0; i < width; i += 2) {
        float2 zf = __half22float2(__half2{data[i], data[i+1]});
        __half2 z = __float22half2_rn(zf * scale);
        data[i] = z.x;
        data[i+1] = z.y;
      }
    } else {
      #pragma unroll
      for (int i = 0; i < width; ++i)
        data[i] = __float2half_rn(__half2float(data[i]) * scale);
    }
    return *this;
  }

  __device__ inline float sum_squares() const {
    float result = 0.0f;
    if constexpr (width % 2 == 0) {
      #pragma unroll
      for (int i = 0; i < width; i += 2) {
        float2 z = __half22float2(__half2{data[i], data[i+1]});
        result += z.x * z.x + z.y * z.y;
      }
    } else {
      #pragma unroll
      for (int i = 0; i < width; ++i) {
        float x = __half2float(data[i]);
        result += x * x;
      }
    }
    return result; 
  }
};

/* Function overload in the case of FP16 tensors.
   Additional optimizations we can make in this case are
   packed and vectorized operations, which help with the
   memory latency bottleneck. */
template<typename scalar_t, int width>
__global__ std::enable_if_t<
  (width > 0) && std::is_same_v<scalar_t, c10::Half>>
fused_add_rms_norm_kernel(
  c10::Half* __restrict__ input,           // [..., hidden_size]
  c10::Half* __restrict__ residual,        // [..., hidden_size]
  const c10::Half* __restrict__ weight,    // [hidden_size]
  const float epsilon,
  const int num_tokens,
  const int hidden_size)
{
  // Ensures reinterpret_cast does not mutate address for alignment reasons
  static_assert(alignof(c10::Half) == alignof(_halfVec<width>));
  // Sanity checks on our vector struct and type-punned pointer arithmetic
  static_assert(std::is_pod_v<_halfVec<width>>);
  static_assert(sizeof(_halfVec<width>) == sizeof(c10::Half) * width);
  const int vec_hidden_size = hidden_size / width;
  __shared__ float s_variance;
  float variance = 0.0f;
  /* These and the argument pointers are all declared `restrict` as they are
     not aliased in practice. Argument pointers should not be dereferenced
     in this kernel as that would be undefined behavior */
  auto* __restrict__ input_v = reinterpret_cast<_halfVec<width>*>(input);
  auto* __restrict__ residual_v = reinterpret_cast<_halfVec<width>*>(residual);
  auto* __restrict__ weight_v = reinterpret_cast<const _halfVec<width>*>(weight);

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    int id = blockIdx.x * vec_hidden_size + idx;
    _halfVec<width> temp = input_v[id];
    temp += residual_v[id];
    variance += temp.sum_squares();
    residual_v[id] = temp;
  }
  /* Keep the following if-else block in sync with the
     calculation of max_block_size in fused_add_rms_norm */ 
  if (num_tokens < 256) {
    variance = blockReduceSum<float, 1024>(variance);
  } else variance = blockReduceSum<float, 256>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    int id = blockIdx.x * vec_hidden_size + idx;
    _halfVec<width> temp = residual_v[id];
    temp *= s_variance;
    temp *= weight_v[idx];
    input_v[id] = temp;
  }
}


/* Generic fused_add_rms_norm_kernel
   The width field is not used but necessary for the correct
   overloading to occur in the FP16 case.
 */
template<typename scalar_t, int width>    // width is not used in this overload
__global__ void fused_add_rms_norm_kernel(
  scalar_t* __restrict__ input,           // [..., hidden_size]
  scalar_t* __restrict__ residual,        // [..., hidden_size]
  const scalar_t* __restrict__ weight,    // [hidden_size]
  const float epsilon,
  const int num_tokens,
  const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    scalar_t z = input[blockIdx.x * hidden_size + idx];
    z += residual[blockIdx.x * hidden_size + idx];
    float x = (float) z;
    variance += x * x;
    residual[blockIdx.x * hidden_size + idx] = z;
  }
  /* Keep the following if-else block in sync with the
     calculation of max_block_size in fused_add_rms_norm */ 
  if (num_tokens < 256) {
    variance = blockReduceSum<float, 1024>(variance);
  } else variance = blockReduceSum<float, 256>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float) residual[blockIdx.x * hidden_size + idx];
    input[blockIdx.x * hidden_size + idx] = ((scalar_t) (x * s_variance)) * weight[idx];
  }
}

} // namespace vllm

void rms_norm(
  torch::Tensor& out,      // [..., hidden_size]
  torch::Tensor& input,    // [..., hidden_size]
  torch::Tensor& weight,   // [hidden_size]
  float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
    input.scalar_type(),
    "rms_norm_kernel",
    [&] {
      vllm::rms_norm_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        epsilon,
        num_tokens,
        hidden_size);
    });
}

#define LAUNCH_FUSED_ADD_RMS_NORM(width)              \
  VLLM_DISPATCH_FLOATING_TYPES(                       \
    input.scalar_type(),                              \
    "fused_add_rms_norm_kernel",                      \
    [&] {                                             \
      vllm::fused_add_rms_norm_kernel                 \
      <scalar_t, width><<<grid, block, 0, stream>>>(  \
        input.data_ptr<scalar_t>(),                   \
        residual.data_ptr<scalar_t>(),                \
        weight.data_ptr<scalar_t>(),                  \
        epsilon,                                      \
        num_tokens,                                   \
        hidden_size);                                 \
    });

void fused_add_rms_norm(
  torch::Tensor& input,    // [..., hidden_size]
  torch::Tensor& residual, // [..., hidden_size]
  torch::Tensor& weight,   // [hidden_size]
  float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  /* This kernel is memory-latency bound in many scenarios.
     When num_tokens is large, a smaller block size allows
     for increased block occupancy on CUs and better latency
     hiding on global mem ops. */
  const int max_block_size = (num_tokens < 256) ? 1024 : 256;
  dim3 block(std::min(hidden_size, max_block_size));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  /*If the tensor types are FP16, try to use the optimized kernel
    with packed + vectorized ops.
    Max optimization is achieved with a width-8 vector of FP16s
    since we can load at most 128 bits at once in a global memory op.
    However, we have to narrow the vectors if the hidden_size does
    not divide 8.
    
    Specifically, assuming hidden-size does not divide 8:
    If the hidden_size divides 4, we can use a width-4 vector.
    If the hidden_size divides 2 or 6, we can use a width-2
      vector.
    If the hidden_size is odd, we can only use a width-1 vector
      which provides no benefit over the base implementation
      => we do not use the optimized kernel, which is signified
      by setting width = 0.
   */
  switch (hidden_size % 8) {
    case 0:
      LAUNCH_FUSED_ADD_RMS_NORM(8);
      break;
    case 2:
      [[fallthrough]];
    case 6:
      LAUNCH_FUSED_ADD_RMS_NORM(2);
      break;
    case 4:
      LAUNCH_FUSED_ADD_RMS_NORM(4);
      break;
    default:
      LAUNCH_FUSED_ADD_RMS_NORM(0);
      break;
  }
}
#undef _FUSED_RMS_MAX_BLOCKSIZE